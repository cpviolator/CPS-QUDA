// includes CUDA Runtime
#include <hip/hip_runtime.h>

//#include <cuda_functions.h>
//#include <cuda_runtime_api.h>

//input A,B; output Tr(A*Bdag)
__global__ void tr_ABdag(double *re_tr_arr, double *A, double *Bdag) {
  
  int snk_idx_vol = threadIdx.x;  //Sink index
  
  double A_arr[288];
  for(int a=0; a<288; a++) A_arr[a] = A[288*(snk_idx_vol) + a];
  

  // Perform trace sum.
  //
  // Tr(ABdag) = A_ab * Bdag_ba 
  //
  // N.B. Bdag enters the function as B. We perform
  // conjugation by manipulating B's row and column
  // indices and multiplying the imaginary elements
  // of B by -1.0 as required.
  // This has the neat result:
  // Tr(AG5_BdagG5) = sum_n sum_m a_nm*b_nm
  
  int s1 = 0;
  int c1 = 0;
  int s2 = 0;
  int c2 = 0;
  int sc_idx = 0;
  
  for(s1=0;s1<4;s1++)
    for(c1=0;c1<3;c1++)
      for(s2=0;s2<4;s2++)
 	for(c2=0;c2<3;c2++) {
 	  //REAL[(reA + im1A*i)(re2B - imB*i)] = (reAreB + imAimB)                                             
 	  sc_idx = 2*(c2 + (3*s2) + (12*c1) + (36*s1));
	  
	  re_tr_arr[snk_idx_vol] += A_arr[sc_idx] * A_arr[sc_idx] + A_arr[sc_idx+1]*A_arr[sc_idx+1];
 	}
}
